#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvbench/nvbench.cuh>
#include <nvbench/main.cuh>
#include <sha256.cuh>
#include <string>
#include <vector>

 __global__ void sha256d_kernel(unsigned char *plaintext, size_t len, unsigned char *digest) {
    bulldozer::cuda::sha256d(plaintext, len, digest);
}

void sha256d_benchmark(nvbench::state& state) {
    const auto plaintext = state.get_string("message");
    unsigned char *d_plaintext = nullptr;
    hipMalloc(&d_plaintext, plaintext.size());
    hipMemcpy(d_plaintext, plaintext.data(), plaintext.size(), hipMemcpyHostToDevice);

    unsigned char *d_digest = nullptr;
    hipMalloc(&d_digest, 32);

    state.exec([=](nvbench::launch& launch) {
        sha256d_kernel<<<1, 1, 0, launch.get_stream()>>>(d_plaintext, plaintext.size(), d_digest);
    });

    hipFree(d_plaintext);
    hipFree(d_digest);
}

NVBENCH_BENCH(sha256d_benchmark)
        .set_name("sha256d_benchmark")
        .add_string_axis("message",
                         std::vector<std::string>{"abc",
                                                  "abcde",
                                                  "Lorem ipsum dolor sit amet, consectetur adipiscing duis.",
                                                  "Lorem ipsum dolor sit amet, consectetur adipiscing elit vivamus."});

NVBENCH_MAIN;